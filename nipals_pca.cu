#include "hip/hip_runtime.h"
// C/C++ example for the CUBLAS (NVIDIA)
// implementation of NIPALS-PCA algorithm
//
// M. Andrecut (c) 2008
//
//to compile
//nvcc -O3 nipals_pca.c -lgsl -lgslcblas -lm -lcublas
//
// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

// includes, cuda
#include <hipblas.h>  //DM deprecated

// matrix indexing convention
#define id(m, n, ld) (((n) * (ld) + (m)))

// declarations
int nipals_cublas(int, int, int,
		  double *, double *, double *);

int print_results(int, int, int,
		  double *, double *, double *, double *);

// main

int main(int argc, char** argv) {

  // PCA model: X = T * P’ + R
  
  // input: X, MxN matrix (data)
  // input: M = number of rows in X
  // input: N = number of columns in X
  // input: K = number of components (K<=N)
  
  // output: T, MxK scores matrix
  // output: P, NxN loads matrix
  // output: R, MxN residual matrix

  int M = 1000, m;
  int N = M/2, n;
  int K = 25;
  
  printf("\nProblem dimensions: MxN=%dx%d, K=%d\n", M, N, K);

  // initialize srand and clock
  srand(time(NULL));
  clock_t start=clock();
  double dtime;

  // initialize cublas
  //hipblasStatus_t status;  //dm replace deprecated
  hipblasStatus_t status;
  status = cublasInit();
  
  if(status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }
  
  // initiallize some random test data X
  double *X;
  X = (double*)malloc(M*N * sizeof(X[0]));

  if(X == 0) {
    fprintf(stderr, "! host memory allocation error: X\n");
    return EXIT_FAILURE;
  }
  
  for(m = 0; m < M; m++) {
    for(n = 0; n < N; n++) {
      X[id(m, n, M)] = rand() / (double)RAND_MAX;
    }
  }

  // allocate host memory for T, P, R

  double *T;
  T = (double*)malloc(M*K * sizeof(T[0]));;
  
  if(T == 0) {
    fprintf(stderr, "! host memory allocation error: T\n");
    return EXIT_FAILURE;
  }
  
  double *P;
  P = (double*)malloc(N*K * sizeof(P[0]));;
  
  if(P == 0) { fprintf(stderr, "! host memory allocation error: P\n");
    return EXIT_FAILURE;
  }
  
  double *R;
  R = (double*)malloc(M*N * sizeof(R[0]));;

  if(R == 0) {
    fprintf(stderr, "! host memory allocation error: R\n");
    return EXIT_FAILURE;
  }
  
  dtime = ((double)clock() - start)/CLOCKS_PER_SEC;
  printf("\nTime for data allocation: %f\n", dtime);

  // call nipals_cublas()
  start=clock();
  memcpy(R, X, M*N * sizeof(X[0]));
  nipals_cublas(M, N, K, T, P, R);
  dtime = ((double)clock() - start)/CLOCKS_PER_SEC;

  printf("\nTime for NIPALS-PCA computation on device: %f\n", dtime);
  print_results(M, N, K, X, T, P, R);

  // memory clean up
  free(R);
  free(P);
  free(T);
  free(X);

  // shutdown
  status = cublasShutdown();

  if(status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "! cublas shutdown error\n"); return EXIT_FAILURE;
  }
  
  if(argc <= 1 || strcmp(argv[1], "-noprompt")) {
    printf("\nPress ENTER to exit...\n"); getchar();
  }
  return EXIT_SUCCESS;
}

int nipals_cublas(int M, int N, int K, double *T, double *P, double *R)
{
  // PCA model: X = T * P’ + R
  
  // input: X, MxN matrix (data)
  // input: M = number of rows in X
  // input: N = number of columns in X (N<=M)
  // input: K = number of components (K<N)

  // output: T, MxK scores matrix
  // output: P, NxK loads matrix
  // output: R, MxN residual matrix
  
  // CUBLAS error handling
  //hipblasStatus_t status;   //dm replace deprecated
  hipblasStatus_t status;

  // maximum number of iterations
  int J = 10000;

  // max error
  double er = 1.0e-7;

  int k, n, j;

  // transfer the host matrix X to device matrix dR
  double *dR = 0;

  status = cublasAlloc(M*N, sizeof(dR[0]), (void**)&dR);
  
  if(status != HIPBLAS_STATUS_SUCCESS) {
    fprintf (stderr, "! device memory allocation error (dR)\n");
    return EXIT_FAILURE;
  }
  
  status = hipblasSetMatrix(M, N, sizeof(R[0]), R, M, dR, M);
  
  if(status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "! device access error (write dR)\n");
    return EXIT_FAILURE; }
    
  // allocate device memory for T, P
  double *dT = 0;
  
  status = cublasAlloc(M*K, sizeof(dT[0]), (void**)&dT);

  if(status != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "! device memory allocation error (dT)\n");
    return EXIT_FAILURE;
  }
  
  double *dP = 0;
  status = cublasAlloc(N*K, sizeof(dP[0]), (void**)&dP);
  
  if(status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "! device memory allocation error (dP)\n");
    return EXIT_FAILURE;
  }
  
  // mean center the data

  double *dU = 0;

  status = cublasAlloc(M, sizeof(dU[0]), (void**)&dU);
  
  if(status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "! device memory allocation error (dU)\n");
    return EXIT_FAILURE;
  }
  hipblasDcopy(M, &dR[0], 1, dU, 1);
  
  for(n=1; n<N; n++) {
    hipblasDaxpy(M, 1.0, &dR[n*M], 1, dU, 1);
  }
  
  for(n=0; n<N; n++) {
    hipblasDaxpy(M, -1.0/N, dU, 1, &dR[n*M], 1);
  }

  double a, b;
  for(k=0; k<K; k++) {
    
    hipblasDcopy(M, &dR[k*M], 1, &dT[k*M], 1);
    
    a = 0.0;

    for(j=0; j<J; j++) {
      
      hipblasDgemv('t', M, N, 1.0, dR, M, &dT[k*M], 1, 0.0, &dP[k*N], 1);
      
      hipblasDscal(N, 1.0/hipblasDnrm2(N, &dP[k*N], 1), &dP[k*N], 1);
      
      hipblasDgemv('n', M, N, 1.0, dR, M, &dP[k*N], 1, 0.0, &dT[k*M], 1);

      b = hipblasDnrm2(M, &dT[k*M], 1);
      
      if(fabs(a - b) < er*b) break;
      
      a = b;
    }
    
    hipblasDger(M, N, -1.0, &dT[k*M], 1, &dP[k*N], 1, dR, M);
    
  }
  
  // transfer device dT to host T
  hipblasGetMatrix(M, K, sizeof(dT[0]), dT, M, T, M);

  // transfer device dP to host P
  hipblasGetMatrix(N, K, sizeof(dP[0]), dP, N, P, N);

  // transfer device dR to host R
  hipblasGetMatrix(M, N, sizeof(dR[0]), dR, M, R, M);

  // clean up memory
  status = cublasFree(dP);
  status = cublasFree(dT);
  status = cublasFree(dR);
  
  return EXIT_SUCCESS;
}

int print_results(int M, int N, int K,
		  double *X, double *T, double *P, double *R)
{
  int m, n, k;

  // If M < 13 print the results on screen

  if(M > 12)
    return EXIT_SUCCESS;

  printf("\nX\n");
  
  for(m=0; m<M; m++) {
    for(n=0; n<N; n++) {
      printf("%+f ", X[id( m, n,M)]);
    }
    printf("\n");
  }
  printf("\nT\n");

  
  for(m=0; m<M; m++) {
    for(n=0; n<K; n++) {
      printf("%+f ", T[id(m, n, M)]);
    }
    printf("\n");
  }
  
  double a;

  printf("\nT’ * T\n");
  
  for(m = 0; m<K; m++) {
    for(n=0; n<K; n++) {
      a=0;

      for(k=0; k<M; k++) {
	a = a + T[id(k, m, M)] * T[id(k, n, M)];
      }
      printf("%+f ", a);
    }
    printf("\n");
  }
  printf("\nP\n");

  for(m=0; m<N; m++) {
    for(n=0; n<K; n++) {
      printf("%+f ", P[id(m, n, N)]);
    }
    printf("\n");
  }
  printf("\nP’ * P\n");
  
  for(m = 0; m<K; m++) {
    for(n=0; n<K; n++) {
      a=0;
      
      for(k=0; k<N; k++) {
	a = a + P[id(k, m, N)] * P[id(k, n, N)];
      }
      
      printf("%+f ", a);
    }
    printf("\n");
  }
  printf("\nR\n");

  for(m=0; m<M; m++) {
    for(n=0; n<N; n++) {
      printf("%+f ", R[id( m, n,M)]);
    }
    printf("\n");
  }
  return EXIT_SUCCESS;
}
